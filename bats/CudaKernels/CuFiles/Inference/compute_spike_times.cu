
#include <hip/hip_runtime.h>
#define INFINITY __int_as_float(0x7f800000)

extern "C" {
    __device__ void get_sample_params(const float **spike_times,
                                      const float **exp_tau_s,
                                      const float **exp_tau,
                                      const float **spike_weights,
                                      int n_neurons, int sample_idx, int neuron_idx, int max_n_pre_spike) {
        int sample_start_idx = sample_idx * max_n_pre_spike;


        *spike_times += sample_start_idx;
        *exp_tau_s += sample_start_idx;
        *exp_tau += sample_start_idx;
        *spike_weights += (sample_idx * n_neurons + neuron_idx) * max_n_pre_spike;
    }

    __device__ void get_neuron_results(int **n_spikes,
                                       float **a,
                                       float **x,
                                       float **spike_times,
                                       float **post_exp_tau,
                                       int n_neurons, int sample_idx, int neuron_idx, int max_n_post_spike) {
        int sample_neuron_idx = (sample_idx * n_neurons + neuron_idx);
        int res_start_idx = sample_neuron_idx * max_n_post_spike;

        *n_spikes += sample_neuron_idx;
        *a += res_start_idx;
        *x += res_start_idx;
        *spike_times += res_start_idx;
        *post_exp_tau += res_start_idx;
    }

    __device__ bool compute_spikes(const float c,
                                   int *n_spikes,
                                   float *a,
                                   float *x,
                                   float *spike_times,
                                   float *post_exp_tau,
                                   float cumul_a,
                                   float *cumul_b,
                                   float last_spike,
                                   float next_spike,
                                   float delta_theta_tau,
                                   float tau,
                                   float max_simulation,
                                   int neuron_idx,
                                   int max_n_post_spike,
                                   int sample_idx) {
        float x_tmp, inside_log, tmp;
        float timestep_freq = 100.;

        // Compute until there is no spike anymore
        while (true) {
            tmp = (*cumul_b) * (*cumul_b) - 4.0 * cumul_a * c;

            if (tmp < 0) // Negative discriminant, no spike --> stop
                return false;
            x_tmp = sqrtf(tmp);
            tmp = x_tmp + (*cumul_b);

            if (tmp == 0.0) // Division per zero, no spike --> stop
                return false;
            inside_log = 2 * cumul_a / tmp;
            if (inside_log < 0) // Negative log, no spike --> stop
                return false;

            tmp = tau * __logf(inside_log);

            // increase firing time to closest time step 
            tmp = ceilf(tmp * timestep_freq) / timestep_freq;

            // check if the spike would also occur at discrete timestep, and if not break
            float potential = - __expf(- tmp/tau) * __expf(- tmp/tau) * cumul_a  + __expf(- tmp/tau) * *cumul_b;
            printf("%e \n", potential);
            if (potential < c)
                return false;

            // Spike time is before the last pre-spike or after the next spike --> stop
            if (tmp <= last_spike || tmp > max_simulation || tmp > next_spike)
                return false;

            // Spike time is valid

            // now update vars used for backprop by reversing the computation
            inside_log = __expf(tmp/tau);
            x_tmp = 2 * cumul_a / inside_log - *cumul_b;
            // TODO should I also change cumul_a, guess so...

            a[*n_spikes] = cumul_a;
            x[*n_spikes] = x_tmp;
            printf("x_tmp=%e\n", x_tmp);
            spike_times[*n_spikes] = tmp;
            last_spike = tmp;
            post_exp_tau[*n_spikes] = inside_log * potential / c;
            *cumul_b -= delta_theta_tau * inside_log;
            (*n_spikes)++;
            if (*n_spikes >= max_n_post_spike) {
                return true;
            }
        }
    }

    __global__ void compute_spike_times_kernel(// Parameters
                                               const float *spike_times,
                                               const float *exp_tau_s,
                                               const float *exp_tau,
                                               const float *spike_weights,
                                               const float c,
                                               float delta_theta_tau,
                                               float tau,
                                               float max_simulation,
                                               int max_n_pre_spike,
                                               int max_n_post_spike,
                                               // Outputs
                                               int *n_spikes,
                                               float *a,
                                               float *x,
                                               float *out_spike_times,
                                               float *post_exp_tau) {
        int n_neurons = gridDim.x;
        int sample_idx = threadIdx.x;
        int neuron_idx = blockIdx.x;


        get_sample_params(&spike_times, &exp_tau_s, &exp_tau, &spike_weights,
                          n_neurons, sample_idx, neuron_idx, max_n_pre_spike);
        get_neuron_results(&n_spikes, &a, &x, &out_spike_times, &post_exp_tau,
                           n_neurons, sample_idx, neuron_idx, max_n_post_spike);

        float cumul_a = 0.0;
        float cumul_b = 0.0;
        float weight;
        int next_i;
        float next_spike;

        for (int i = 0; i < max_n_pre_spike; i++) {
            if (spike_times[i] == INFINITY) // No spike anymore --> stop
                break;
            weight = spike_weights[i];

            cumul_a += weight * exp_tau_s[i];
            cumul_b += weight * exp_tau[i];

            next_i = i + 1;
            if (next_i < max_n_pre_spike)
                next_spike = spike_times[next_i];
            else
                next_spike = INFINITY;

            if (compute_spikes(c, n_spikes, a, x, out_spike_times, post_exp_tau,
                               cumul_a, &cumul_b, spike_times[i], next_spike, delta_theta_tau, tau,
                               max_simulation, neuron_idx, max_n_post_spike, sample_idx))
                break; // Buffer full
        }
    }
}
